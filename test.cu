#include <dlfcn.h>
#include <string.h>
#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    std::cout << "testing cudaMalloc..." << std::endl;
    float *d_x;

    hipMalloc(&d_x, 1024*8*sizeof(float));

    std::cout << "testing cudaFree..." << std::endl;
    hipFree(&d_x);

    return 0;
}